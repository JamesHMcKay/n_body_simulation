#include "hip/hip_runtime.h"
#include <iostream>

#include "gtest/gtest.h"
#include "../include/kernel.cuh"
#include "hip/hip_vector_types.h"

TEST(test_physics, add_acceleration) {
  Physics physics;
  float4 p1 = make_float4(0, 0, 0, 0);
  float4 p2 = make_float4(0, 0, 0, 0);
  float3 a = make_float3(0, 0, 0);
  float3 result = physics.add_acceleration(p1, p2, a);

  EXPECT_EQ(physics.add_acceleration(p1, p2, a).x, make_float3(0, 0, 0).x);
}